#include "ForestFireModel.h"

namespace ForestFire {
	ForestFireModel::ForestFireModel(ForestFireConsts consts, ForestFireDataH data)
			: IterativeModel(consts.Tau), _consts(consts), _data(data) {

		for(size_t y = 0; y < data.dimY(); y++) {
			data.t(0, y) = consts.TemOnBounds;
			data.t(data.dimX() - 1, y) = consts.TemOnBounds;
		}

		for (size_t x = 0; x < data.dimX(); x++) {
			data.t(x, 0) = consts.TemOnBounds;
			data.t(x, data.dimY() - 1) = consts.TemOnBounds;
		}

		sf_h = HostData2D<SweepFactors<float>>(data.dimX(), data.dimY());

		t4CPU = T4<ForestFireDataH>(consts, data);
		gorenieCPU = Gorenie<ForestFireDataH>(consts, data);
		temperatureCPU = Temperature<ForestFireDataH>(consts, data);

		AddCalculationMethod("cpu", std::bind(std::mem_fun(&ForestFireModel::CalculationMethod_CPU), this));
		AddCalculationMethod("gpu", std::bind(std::mem_fun(&ForestFireModel::CalculationMethod_GPU), this));
	}

	ForestFireModel::~ForestFireModel() {
		GpuOff();

		_data.Erase();
		sf_h.Erase();
	}

	std::string ForestFireModel::PrintData() const {
		return "";
	}

	void ForestFireModel::SynchronizeWithGpu() {
		if (isGpuOn())
			_data_dev.PutTo(_data);
	}

	void ForestFireModel::PrepareDataForGpu(const Sonsode::GpuDevice &gpuDevice, size_t orderNumber) {
		sf_d = DeviceData2D<SweepFactors<float>>(gpuDevice, sf_h);
		_data_dev = ForestFireDataD(gpuDevice, _data);

		t4GPU = T4<ForestFireDataD>(_consts, _data_dev);
		gorenieGPU = Gorenie<ForestFireDataD>(_consts, _data_dev);
		temperatureGPU = Temperature<ForestFireDataD>(_consts, _data_dev);
	}

	void ForestFireModel::FreeDataForGpus() {
		_data_dev.Erase();
		sf_d.Erase();
	}

	void ForestFireModel::CalculationMethod_CPU() {
		GpuOff();

		//������������� ����������
		ExplicitGaussSeidel_2D_CPU(t4CPU);
	
		//����� �����������
		ImplicitSweep_2D_CPU(sf_h, temperatureCPU);

		//�������
		if (currentIteration() >= _consts.IterFireBeginNum)
			FullSearch_2D_CPU(gorenieCPU);
	}

	void ForestFireModel::CalculationMethod_GPU() {
		GpuOn();

		//������������� �����������
		ExplicitGaussSeidel_2D_GPU_direct(t4GPU);

		//����� �����������
		ImplicitSweep_2D_GPU_lineDivide(sf_d, temperatureGPU);

		//�������
		if (currentIteration() >= _consts.IterFireBeginNum)
			FullSearch_2D_GPU(gorenieGPU);
	}
}